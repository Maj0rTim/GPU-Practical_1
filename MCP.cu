#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <hip/hip_runtime_api.h>

#define ITER 100'000'000


/*

Monte Carlo calculation of PI - An attempt - Timothy Fischer

The following code should estimate the value of pi. It will consistently output a value around 1.57, which is about 
half the value of pi, I'm not sure why this happens. I end up just doubling the estimated value to get a vlaue close to pi.

Some of the ways I have tried to fix/optimize this code is by understanding how the threads and streaming multiprocessors 
work. The GTX 750 TI can have a max of 2048 threads running on each of the five streaming multiprocessors. 
This gives a total of 10 480 threads. If we want to generate close to 100,000,000 numbers, we need each thread to 
generate 9765 random numbers. This gives us 10480*9765=99,993,600 random numbers for the estimation. 
The block and grid size have been set to 160 x 64 respectivly to ensure we utilize the total number of threads available.

Achieved occupancy: 84.6


*/


__global__ void cudaRand(double *device_counts)
{
    int i, local_count = 0;
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);

    hiprandState state;
    hiprand_init(35791246, idx, 0, &state);

    for (i=0; i<9765; i++)
    {
        double x = hiprand_uniform_double(&state);
        double y = hiprand_uniform_double(&state);
        if ((x * x) + (y * y) <= 1.0) 
        {
            local_count++;
        }
    }
    
    device_counts[idx] = local_count;

}

int main(int argc, char** argv)
{
    // initialise CUDA timing
	float milli;
	hipEvent_t start_kernel, stop_kernel, start_program, stop_program;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
    hipEventCreate(&start_program);
	hipEventCreate(&stop_program);

    hipEventRecord(start_program); 

    int niter = ITER;
    size_t size = 64*160;
    double *host_counts = new double[size];

    double *device_counts;
    checkCudaErrors(hipMalloc((void**)&device_counts, size * sizeof(double)));


    // Set Kernel Parameters
    dim3 block(64, 1);
    dim3 grid(160, 1, 1);

    // Launch Kernel
    hipEventRecord(start_kernel);  
    checkCudaErrors(hipDeviceSynchronize());
    cudaRand <<< block, grid >>> (device_counts);
    hipEventRecord(stop_kernel);
    checkCudaErrors(hipEventSynchronize(stop_kernel));
    hipEventElapsedTime(&milli, start_kernel, stop_kernel);  

    printf("Double Generator <<<(%d,%d), (%d,%d)>>>\n", grid.x, grid.y,
        block.x, block.y);
    printf("Kernel Execution Time: %f ms\n", milli);
     // Copy final counter values from device to host
    checkCudaErrors(hipMemcpy(host_counts, device_counts, size * sizeof(double), hipMemcpyDeviceToHost));

    double final_count = 0.0;
    for (size_t i = 0; i < size; i++)
        final_count+=host_counts[i];
    
    double pi = (double)final_count / (niter) * 4.0;

    hipEventRecord(stop_program);
    checkCudaErrors(hipEventSynchronize(stop_program));
    hipEventElapsedTime(&milli, start_program, stop_program);  
    printf("Total Program Time: %f ms\n", milli);
    printf("Estimate for PI: %g\n", pi);

    checkCudaErrors(hipFree(device_counts));
    delete[] host_counts;

    return 0;
}